#include "hip/hip_runtime.h"
// matmul_compare.cu
#include "test.h"
#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cblas.h>

// nvcc -o test main.cu -lopenblas

void origin(float* y, const float* q, const float* k, const float* v, const int dim, const int q_head, const int kv_head, const int _pos);
void masked_attention_cpu(float* y, float* q, float* k, float* v, float* scores, int* pos, int dim, int head_num, int seq_q, int seq_kv);
void masked_attention_cuda(
    float* y, 
    float* q, 
    float* k, 
    float* v, 
    float* scores, 
    int* pos, 
    int dim, 
    int head_num,
    int seq_q,
    int seq_kv
);

int main() {
    size_t pos = 5;
    size_t hidden_size = 2048;
    size_t head_dim = 64;
    size_t head_num = 32;

    size_t scores_size = 5 * head_num * 5;

    assert(hidden_size == head_dim*head_num);

    Test testtool;
    testtool.setDevice("cpu");

    float* q = testtool.getArr(hidden_size * pos, true);
    float* k = testtool.getArr(hidden_size * pos, true);
    float* v = testtool.getArr(hidden_size * pos, true);
    float* scores = testtool.getArr(scores_size);
    float* o1 = testtool.getArr(hidden_size*pos);
    float* o2 = testtool.getArr(hidden_size*pos);

    int offset = 0;
    for(int p = 0; p < pos; p++) {
        origin(o1 + offset*hidden_size, q + offset*hidden_size, k, v, head_dim, head_num, head_num, p);
        offset += 1;
    }

    int position[pos];
    position[0] = 0;position[1] = 1;position[2] = 2;position[3] = 3;position[4] = 4;
    masked_attention_cpu(o2, q, k, v, nullptr, position, head_dim, head_num, pos, pos);


    testtool.check(o1, o2, hidden_size * pos, "CPU Attention");

    testtool.setDevice("cuda");

    float* o3 = testtool.get_from_cpu(o2, hidden_size*pos);
    float* o_cuda = testtool.getArr(hidden_size*pos);
    float* q_cuda = testtool.get_from_cpu(q, hidden_size * pos);
    float* k_cuda = testtool.get_from_cpu(k, hidden_size * pos);
    float* v_cuda = testtool.get_from_cpu(v, hidden_size * pos);

    int* p_cuda = (int*)testtool.get_from_cpu((float*)position, pos * sizeof(float)/sizeof(int));

    masked_attention_cuda(o_cuda, q_cuda, k_cuda, v_cuda, nullptr, p_cuda, head_dim, head_num, pos, pos);

    testtool.check(o3, o_cuda, hidden_size * pos, "CUDA Attention");
}

void softmax_cpu(float *x, int n, int batch_size) {
    for(int b = 0; b < batch_size; b++) {
        // 找到输入数组中的最大值，以提高数值稳定性
        float* input = x + b * n;
        float max_val = input[0];
        for(int i = 1; i < n; ++i){
            if(input[i] > max_val){
                max_val = input[i];
            }
        }

        // 计算每个元素的指数值，并累加
        float sum = 0.0f;
        for(int i = 0; i < n; ++i){
            input[i] = std::exp(input[i] - max_val);
            sum += input[i];
        }

        // 将每个指数值除以总和，得到概率分布
        for(int i = 0; i < n; ++i){
            input[i] /= sum;
        }
    }
}

void origin(float* y, const float* q, const float* k, const float* v, const int dim, const int q_head, const int kv_head, const int _pos) {
    int pos = _pos + 1;
    float* score = new float[q_head * pos](); // 置初始值为0，列优先，pos行，q_head列

    int rep = q_head / kv_head;
    int kv_dim = kv_head * dim;

    float scale = 1.0 / std::sqrt(static_cast<float>(dim));
    for(int p = 0; p < pos; p++) {
        for(int hq = 0; hq < q_head; hq++) {
            const float* _q = q + hq * dim;
            const float* _k = k + p * kv_dim + (hq / rep) * dim;
            const int s_index = hq*pos + p;
            for(int d = 0; d < dim; d++) {
                score[s_index] += _q[d] * _k[d];
            }
            score[s_index] *= scale;
        }
    }

    softmax_cpu(score, pos, q_head);

    std::memset(y, 0, dim * q_head * sizeof(float));

    for(int hq = 0; hq < q_head; hq++) {
        float* _s = score + hq * pos;
        float* _y = y + hq * dim;
        for(int p = 0; p < pos; p++) {
            const float* _v = v + p * kv_dim + (hq / rep) * dim;
            for(int d = 0; d < dim; d++) {
                _y[d] += _s[p] * _v[d];
            }
        }
    }

    delete score;
}

inline float dot(float* a, float* b, size_t size) {
    return cblas_sdot(size, a, 1, b, 1);
}

inline void scale(float* a, float alpha, size_t size) {
    cblas_sscal(size, alpha, a, 1);
}

// Y = alpha * X + Y
inline void add(float* y, float* x, size_t size, float alpha = 1) {
    cblas_saxpy(size, alpha, x, 1, y, 1);
}

void masked_attention_cpu(float* y, float* q, float* k, float* v, float* scores, int* pos, int dim, int head_num, int seq_q, int seq_kv) {
    bool hasvalue = true;
    if(scores == nullptr) {
        scores = new float[seq_kv * head_num];
        hasvalue = false;
    }

    std::memset(y, 0, dim * head_num * seq_q * sizeof(float));

    float scale_ = 1.0 / std::sqrt(static_cast<float>(dim));

    int kv_num_ = seq_kv - seq_q;
    for(int i_q = 0; i_q < seq_q; i_q++) {
        kv_num_++;
        float* q_ = q + i_q * dim * head_num;
        float* y_ = y + i_q * dim * head_num;
        for(int i_kv = 0; i_kv < kv_num_; i_kv++) {
            float* k_ = k + i_kv * dim * head_num;
            for(int h = 0; h < head_num; h++) {
                scores[i_kv + h*kv_num_] = dot(q_ + h*dim, k_ + h*dim, dim);
            }
        }
        scale(scores, scale_, kv_num_*head_num);
        softmax_cpu(scores, kv_num_, head_num);

        for(int i_kv = 0; i_kv < kv_num_; i_kv++) {
            float* v_ = v + i_kv * dim * head_num;
            for(int h = 0; h < head_num; h++) {
                add(y_ + h*dim, v_ + h*dim, dim, scores[i_kv + h*kv_num_]);
            }
        }
    }

    if(!hasvalue) delete scores;
}



// q [seq_q,  head_num, dim]
// k [seq_kv, head_num, dim]
// kernel<<<(seq_kv, head_num), (seq_q)>>>
// scores [seq_q, head_num, seq_kv]
__global__ void compute_masked_scores_kernel(
    float* scores,
    float* __restrict__ q,
    float* __restrict__ k_cache,
    int* q_pos,
    int dim,
    float  scale
) {
    int kv_id = blockIdx.x;      // gridDim.x = seq_kv
    int head_id = blockIdx.y;    // gridDim.y = head_num
    int q_id = threadIdx.x;      // blockDim.x = seq_q

    int kv_num = gridDim.x;      // seq_kv
    int head_num = gridDim.y;    // head_num

    int pos = q_pos[q_id];

    float sum = 0.0f;
    #pragma unroll
    for(int i = 0; i < dim; i++) {
        sum += q[q_id * head_num * dim + head_id*dim + i] * k_cache[kv_id*head_num*dim + head_id*dim + i];
    }

    if(kv_id <= pos) {
        scores[q_id*head_num*kv_num + head_id*kv_num + kv_id] = sum * scale;
    } else {
        scores[q_id*head_num*kv_num + head_id*kv_num + kv_id] = -INFINITY;
    }
}

// o      [seq_q, head_num, dim]
// scores [seq_q, head_num, seq_kv]
// kernel<<<(seq_q), (head_num)>>>
__global__ void compute_masked_output_kernel(
    float* o,
    float* v_cache,
    float* scores,
    int kv_num,
    int dim
) {
    int head_num = blockDim.x;

    int h_id = threadIdx.x;
    int q_id = blockIdx.x;
    

    for(int i = 0; i < kv_num; i++) {
        float s = scores[q_id*head_num*kv_num + h_id*kv_num + i];
        #pragma unroll
        for(int d = 0; d < dim; d++) {
            o[q_id*head_num*dim + h_id*dim + d] += s * v_cache[i*head_num*dim + h_id*dim + d];
        }
    }

}

__global__ void softmax_gpu(float *__restrict__ x, int size) {
    int tid = threadIdx.x;
    int block_size = blockDim.x;

    int batch_idx = blockIdx.y;
    int idx = batch_idx * size;

    x += idx;

    // 找到最大值（用于数值稳定性）
    float max_val = -FLT_MAX;
    for (int i = tid; i < size; i += block_size) {
        if (x[i] > max_val) {
            max_val = x[i];
        }
    }

    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float shared_max;

    float max_result = BlockReduce(temp_storage).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0) {
        shared_max = max_result;
    }
    __syncthreads();
    max_val = shared_max;

    // 计算指数和总和
    float sum = 0.0f;
    for (int i = tid; i < size; i += block_size) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }

    sum = BlockReduce(temp_storage).Sum(sum);
    if (threadIdx.x == 0) {
        shared_max = sum;
    }
    __syncthreads();
    sum = shared_max;

    // 归一化
    for (int i = tid; i < size; i += block_size) {
        x[i] /= sum;
    }
}


void masked_attention_cuda(
    float* y, 
    float* q, 
    float* k, 
    float* v, 
    float* scores, 
    int* pos, 
    int dim, 
    int head_num,
    int seq_q,
    int seq_kv
) {
    float scale = 1.0f / std::sqrt(static_cast<float>(dim));
    bool hasvalue = true;
    if(scores == nullptr) {
        hasvalue = false;
        hipError_t err = hipMalloc((void**)&scores, seq_kv*head_num*dim*sizeof(float));
        if (err != hipSuccess) {
            std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
            return;
        }
    }

    compute_masked_scores_kernel<<<dim3(seq_kv, head_num), dim3(seq_q)>>>(scores, q, k, pos, dim, scale);

    softmax_gpu<<<dim3(1, seq_q * head_num), dim3(1024)>>>(scores, seq_kv);

    compute_masked_output_kernel<<<dim3(seq_q), dim3(head_num)>>>(y, v, scores, seq_kv, dim);

    if(!hasvalue) hipFree(scores);
}


